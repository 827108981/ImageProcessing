#include "SVD.h"

SVDT::SVDT(const int m, const int n)
{
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
	cublas_status = hipblasCreate(&cublasH);
	assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

	cudaStat1 = hipMalloc((void **)&d_A, sizeof(float) * m * n);   // Storing input image
    cudaStat2 = hipMalloc((void **)&d_U, sizeof(float) * m * m);    // Left singular matrix
	cudaStat3 = hipMalloc((void **)&d_S, sizeof(float) * m * n);   // singular - values
	cudaStat4 = hipMalloc((void **)&d_VT, sizeof(float) * n * n);
	cudaStat5 = hipMalloc((void**)&devInfo, sizeof(int));

	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	assert(hipSuccess == cudaStat4);
	assert(hipSuccess == cudaStat5);

	cusolver_status = hipsolverDnSgesvd_bufferSize(
			cusolverH,
			m,
			n,
			&lwork);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
	cudaStat3 = hipMalloc((void**)&r_work, sizeof(float)*lwork);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat3);
}

SVDT::~SVDT()
{
	//cusolver_status = hipsolverDnDestroy(cusolverH);
	//assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    if (d_A)
		hipFree(d_A);
	if (d_U)
		hipFree(d_U);
	if (d_S)
		hipFree(d_S);
	if (d_VT)
		hipFree(d_VT);
	if (devInfo)
		hipFree(devInfo);
	if (d_work)
		hipFree(d_work);
	if (r_work)
		hipFree(r_work);

	if (cublasH)
		hipblasDestroy(cublasH);
	if (cusolverH)
		hipsolverDnDestroy(cusolverH);
}

void SVDT::SVDcompute(int m, int n, int lda, int ldu, int ldvt, const float *A, float *U, float *S, float *VT)
{
	cudaStat1 = hipMemcpy(d_A, A, sizeof(float)*lda*n, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
    hipEventRecord(startEvent);
	//compute SVD
	cusolver_status = hipsolverDnSgesvd(
		cusolverH,
		'A',
		'A',
		m,
		n,
		d_A,
		lda,
		d_S,
		d_U,
		ldu,
		d_VT,
		ldvt,
		d_work,
		lwork,
		r_work,
		devInfo
		);
	hipEventRecord(stopEvent);
	hipEventSynchronize(stopEvent);
	float elapsedTime = 0.0f;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
	cout << "SVD Time: " << elapsedTime << " ms." << endl;

	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	//check if SVD is good or not
	cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(0 == info_gpu);

	//copy U S and VT to host
	cudaStat1 = hipMemcpy(U, d_U, sizeof(float)*ldu*m, hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(S, d_S, sizeof(float)*n, hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(VT, d_VT, sizeof(float)*ldvt*n, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	//hipDeviceReset();
}
