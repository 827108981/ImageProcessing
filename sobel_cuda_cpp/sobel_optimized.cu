#include "hip/hip_runtime.h"
#include "sobel.h"

// declare texutre reference globally
texture<float, hipTextureType2D, hipReadModeElementType> texIn;

__constant__ float *sobelX;
__constant__ float *sobelY;

__global__ void SobelKernel(int row, int col, float *imgOut )
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//int index = x + y * blockDim.x * gridDim.x;
	int index = x + y * col;

	if(x < col && y < row)
	{
		// do the calculation parallelized
		float lu = tex2D(texIn, x - 1, y - 1);
	    float lm = tex2D(texIn, x, y - 1);
		float lb = tex2D(texIn, x + 1, y - 1);

		float mu = tex2D(texIn, x - 1, y);
		float mm = tex2D(texIn, x, y);
		float mb = tex2D(texIn, x + 1, y);

		float ru = tex2D(texIn, x - 1, y + 1);
		float rm = tex2D(texIn, x , y +1);
		float rb = tex2D(texIn, x + 1, y + 1);

		float tX = 0, tY = 0, T = 0;
		tX = (-1) * lu + lb - 2 * mu + 2 * mb - ru + rb;
		tY = lu + 2 * lm + lb - ru - 2 * rm - rb;

		T = sqrt(tX * tX + tY * tY);

		// Now the 200 is the threshold value
		if(T > 100)
			imgOut[index] = 255;
		else if(T < 50)
			imgOut[index] = 0;
		else
			imgOut[index] = T;
	}
}

__global__ void copy_texture_kernel(float *iptr)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockIdx.x * blockDim.x;

	float c = tex2D(texIn, x, y);
	if(c != 0)
		iptr[offset] = c;
}

// float *A is the image data stored on host
void mySobel::SobelCompute(float *A, float *B)
{
	hipError_t cudaState_1 = hipSuccess;

	cout << "row = " << row << "col = " << col << endl;

	// declare sobel operator matrix
	float temp_sobelX[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
	float temp_sobelY[9] = {-1, -1, -1, 0, 0, 0, 1, 2, 1};
	cout << "sizeof(*temp_sobleX) : " << sizeof(*temp_sobelX) << endl;
	hipMemcpyToSymbol(HIP_SYMBOL(sobelX), temp_sobelX, sizeof(float) * 9);  // no need to malloc memory for constant variables
	hipMemcpyToSymbol(HIP_SYMBOL(sobelY), temp_sobelY, sizeof(float) * 9);


	// measure the performance with the help of envent
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *imgIn_dev, *imgOut_dev;

	const int SIZE = row * col * sizeof(float);

	cudaState = hipMalloc((void**)&imgIn_dev, SIZE);
	assert(cudaState == hipSuccess);
	cudaState = hipMalloc((void **)&imgOut_dev, SIZE);
	assert(cudaState == hipSuccess);
	hipMemset(imgOut_dev, 0, SIZE);

	// copy the image data from host to device
	cudaState_1 = hipMemcpy(imgIn_dev, A, SIZE, hipMemcpyHostToDevice);
	assert(cudaState_1 == hipSuccess);

	// bind the cuda memory to the texture memory

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	// create 2D array
	hipArray* imgArray;
	hipMallocArray(&imgArray, &desc, col, row);
	// copy  to device memory some data locate at input image
	hipMemcpyToArray(imgArray, 0, 0, A, SIZE, hipMemcpyHostToDevice);
	// set texture parameters;
	texIn.addressMode[0] = hipAddressModeWrap;
	texIn.addressMode[1] = hipAddressModeWrap;
	texIn.filterMode = hipFilterModeLinear;
	texIn.normalized = false;
	hipBindTextureToArray(texIn, imgArray, desc);
	//hipBindTexture2D(NULL, texIn, imgArray, desc, col, row, sizeof(float) * col);
	//	hipBindTexture(NULL, texOut, imgOut_dev, SIZE);

	// Call the kernel function
	dim3 thread(16, 16);    // 16 * 16 threads per block
	dim3 grid((row + 15)/16, (col + 15)/ 16);

	hipEventRecord(start, 0);
	SobelKernel<<<grid, thread>>>(row, col, imgOut_dev);
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "Sobel calculated on GPU cost " << elapsedTime << " ms" << endl;

	// copy result back to host
	hipMemcpy(B, imgOut_dev, SIZE, hipMemcpyDeviceToHost);
	// for test
	for(int i = 0; i < 10; i++)
	{
		cout << B[i] << endl;
	}

	// clean up memory allocated on The GPU
	hipUnbindTexture(texIn);
	//hipUnbindTexture(TexOut);
	hipFreeArray(imgArray);
	hipFree(imgIn_dev);
	hipFree(imgOut_dev);
	// destroy the event variables
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
