#include "hip/hip_runtime.h"
#include "gaussfilter.h"

#define cudaErrorCheck(err) __checkCUDAError(err, __FILE__, __LINE__)

#define BLOCKSIZE 16
#define FILTERRAD 5
#define TILE_WIDTH (BLOCKSIZE + 2 * FILTERRAD)

#define INDX(r, c, w) ((r) * (w) + (c))

#define FILTERSIZE_ (11*11)

__constant__ float d_filter_const_[FILTERSIZE_];

// declare the texture memory
texture<float, hipTextureType2D, hipReadModeElementType> texIn;

inline void __checkCUDAError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        cout << err << " in " << file << " at " << line << " line.";
        exit(EXIT_FAILURE);
    }
}

// filter is on constant memory
__global__ void
gaussfilterCon_kernel(float *d_imgOut, float *d_imgIn, int wid, int hei, const float *__restrict__ d_filter,
                      int filterW) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx > wid || idy > hei)
        return;

    int filterR = (filterW - 1) / 2;

    float val = 0.f;

    for (int fr = -filterR; fr <= filterR; ++fr)           // row
        for (int fc = -filterR; fc <= filterR; ++fc)      // col
        {
            int ir = idy + fr;
            int ic = idx + fc;

            if ((ic >= 0) && (ic <= wid - 1) && (ir >= 0) && (ir <= hei - 1))
                val += d_imgIn[INDX(ir, ic, wid)] * d_filter_const_[INDX(fr + filterR, fc + filterR, filterW)];
        }
    d_imgOut[INDX(idy, idx, wid)] = val;

}

__global__ void
gaussfilterGlo_kernel(float *d_imgOut, float *d_imgIn, int wid, int hei, const float *__restrict__ d_filter,
                      int filterW) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx > wid || idy > hei)
        return;

    int filterR = (filterW - 1) / 2;

    float val = 0.f;

    for (int fr = -filterR; fr <= filterR; ++fr)           // row
        for (int fc = -filterR; fc <= filterR; ++fc)      // col
        {
            int ir = idy + fr;
            int ic = idx + fc;

            if ((ic >= 0) && (ic <= wid - 1) && (ir >= 0) && (ir <= hei - 1))
                val += d_imgIn[INDX(ir, ic, wid)] * d_filter[INDX(fr + filterR, fc + filterR, filterW)];
        }
    d_imgOut[INDX(idy, idx, wid)] = val;
}

__global__ void gaussfilterTex_kernel(float *d_imgOut, float *d_imgIn, int wid, int hei, float *d_filter, int filterW) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx > wid || idy > hei)
        return;

    int filterR = (filterW - 1) / 2;

    float val = 0.f;

    for (int fr = -filterR; fr <= filterR; fr++)
        for (int fc = -filterR; fc <= filterR; fc++) {
            val += tex2D(texIn, idy + fr, idx + fc) * d_filter[INDX(fr + filterR, fc + filterR, filterW)];
        }

    d_imgOut[INDX(idy, idx, wid)] = val;
}

/*
__global__ void gaussfilterSha_kernel(float *d_imgOut, float *d_imgIn, int wid, int hei, float *d_filter, int filterW)
{
    int x0 = threadIdx.x;
    int y0 = threadIdx.y;

    int idx = x0 + blockDim.x * blockIdx.x;
    int idy = y0 + blockDim.y * blockIdx.y;

    if(idx >= wid || idy >= hei)
        return ;

    __shared__ float shareMem[TILE_WIDTH * TILE_WIDTH];

    int x, y;

    // case 1 : upper left
    x = idx - FILTERRAD;
    y = idx - FILTERRAD;
    if(x < 0 || y < 0)
        shareMem[INDX(y0, x0, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0, x0, TILE_WIDTH)] = d_imgIn[INDX(y, x, wid)];
    // shareMem[INDX(y0, x0, TILE_WIDTH)] = d_imgIn[INDX(idy, idx, wid) - FILTERRAD - FILTERRAD * wid]

    // case 2 : upper right
    x = idx + FILTERRAD;
    y = idy - FILTERRAD;
    if(x >= wid || y < 0)
        shareMem[INDX(y0, x0+2*FILTERRAD, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0, x0+2*FILTERRAD, TILE_WIDTH)] = d_imgIn[INDX(y, x, wid)];
    //  shareMem[INDX(y0, x0+FILTERRAD, TILE_WIDTH)] = d_imgIn[INDX(idy, idx, wid) + FILTERRAD - FILTERRAD * wid]

    // case 3 : lower left
    x = idx - FILTERRAD;
    y = idy + FILTERRAD;
    if(x < 0 || y >= hei)
        shareMem[INDX(y0+2*FILTERRAD, x0, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0+2*FILTERRAD, x0, TILE_WIDTH)] = d_imgIn[INDX(y, x, wid)];
    //  shareMem[INDX(y0+FILTERRAD, x0, TILE_WDITH)] = d_imgIn[INDX(idy, idx, wid) - FILTERRAD + FILTERRAD * wid]

    // case 4 : lower right
    x = idx + FILTERRAD;
    y = idy + FILTERRAD;
    if(x >= wid || y >= hei)
        shareMem[INDX(y0+2*FILTERRAD, x0+2*FILTERRAD, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0+2*FILTERRAD, x0+2*FILTERRAD, TILE_WIDTH)] = d_imgIn[INDX(y, x, wid)];
    //  shareMem[INDX(y0+FILTERRAD, x0+FILTERRAD, TILE_WIDTH)] = d_imgIn[INDX(idy, idx, wid) + FILTERRAD + FILTERRAD * wid]

    __syncthreads();

    float val = 0.f;

    for(int fr = 0; fr <= filterW; fr++)
        for(int fc = 0; fc <= filterW; fc++)
        {
            val += shareMem[INDX(y0+fr, x0 + fc, TILE_WIDTH)] * d_filter[INDX(fr, fc, filterW)];
        }

    //assert(val > 0);
    d_imgOut[INDX(idy, idx, wid)] = val;
}
*/

__global__ void gaussfilterSha_kernel(float *d_imgOut, float *d_imgIn, int wid, int hei, float *d_filter, int filterW) {
    int x0 = threadIdx.x;
    int y0 = threadIdx.y;

    int idx = blockDim.x * blockIdx.x + x0;
    int idy = blockDim.y * blockIdx.y + y0;

    if (idx >= wid || idy >= hei)
        return;

    int filterR = (filterW - 1) / 2;

    __shared__ float shareMem[TILE_WIDTH * TILE_WIDTH];

    int x, y;

    // case 1 : upper left
    x = idx - FILTERRAD;
    y = idy - FILTERRAD;
    if (x < 0 || y < 0)
        shareMem[INDX(y0, x0, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0, x0, TILE_WIDTH)] = d_imgIn[INDX(idy, idx, wid) - FILTERRAD - INDX(FILTERRAD, 0, wid)];

    // case 2 : upper right
    x = idx + FILTERRAD;
    y = idy - FILTERRAD;
    if (x >= wid || y < 0)
        shareMem[INDX(y0, x0 + 2*FILTERRAD, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0, x0 + 2*FILTERRAD, TILE_WIDTH)] = d_imgIn[INDX(idy, idx, wid) + FILTERRAD -
                                                               INDX(FILTERRAD, 0, wid)];

    // case 3 : lower left
    x = idx - filterR;
    y = idy + filterR;
    if (x < 0 || y >= hei)
        shareMem[INDX(y0 + 2*filterR, x0, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0 + 2*filterR, x0, TILE_WIDTH)] = d_imgIn[INDX(idy, idx, wid) - FILTERRAD +
                                                               INDX(FILTERRAD, 0, wid)];

    // case 4 : lower right
    x = idx + filterR;
    y = idy + filterR;
    if (x >= wid || y >= hei)
        shareMem[INDX(y0 + 2*FILTERRAD, x0 + 2*FILTERRAD, TILE_WIDTH)] = 0;
    else
        shareMem[INDX(y0 + 2*FILTERRAD, x0 + 2*FILTERRAD, TILE_WIDTH)] = d_imgIn[INDX(idy, idx, wid) + FILTERRAD + INDX(FILTERRAD, 0, wid)];

    __syncthreads();

    // convolution
    float sum = 0.f;

    x = FILTERRAD + threadIdx.x;
    y = FILTERRAD + threadIdx.y;
    for (int i = -FILTERRAD; i <= FILTERRAD; ++i)                       // row
        for (int j = -FILTERRAD; j <= FILTERRAD; ++j)                   // col
            sum += shareMem[INDX(y + j, x + i, TILE_WIDTH)] * d_filter[INDX(j + FILTERRAD, i + FILTERRAD, filterW)];

    d_imgOut[INDX(idy, idx, wid)] = sum;
}

GFilter::GFilter(int wid, int hei, int filterW, float sig) {
    hipError_t cudaState = hipSuccess;
    cudaState = hipMalloc((void **) &d_imgIn_, sizeof(float) * wid * hei);
    assert(cudaState == hipSuccess);
    cudaState = hipMalloc((void **) &d_imgOut_, sizeof(float) * wid * hei);
    assert(cudaState == hipSuccess);
    cudaState = hipMalloc((void **) &d_filter_, sizeof(float) * filterW * filterW);
    assert(cudaState == hipSuccess);

    filterW_ = filterW;
    filterR_ = (filterW - 1) / 2;
    filterSize_ = filterW * filterW;
    sig_ = sig;

    filter_ = new float[filterW * filterW];
}

GFilter::~GFilter() {
    if (filter_)
        delete[] filter_;
    if (d_filter_)
        hipFree(d_filter_);
    if (d_imgOut_)
        hipFree(d_imgOut_);
    if (d_imgIn_)
        hipFree(d_imgIn_);
}

// prepare the gaussian filter
void GFilter::createfilter() {
    //hipError_t cudaState = hipSuccess;

    float val = 0.f;
    float sum = 0.f;

    float sig = 2 * sig_ * sig_;
    for (int i = -filterR_; i <= filterR_; ++i)       // row
    {
        for (int j = -filterR_; j <= filterR_; ++j)   // col
        {
            val = i * i + j * j;
            val = exp(-val / sig) / (sig * PI);
            sum += val;
            int offset = (i + filterR_) * filterW_ + j + filterR_;
            filter_[offset] = val;
        }
    }

    for (int i = 0; i < filterSize_; i++)
        filter_[i] *= 1.0 / sum;

    cudaErrorCheck(hipMemcpy(d_filter_, filter_, sizeof(float) * filterSize_, hipMemcpyHostToDevice));

    // copy data from host to constant memory on device
    cudaErrorCheck(
            hipMemcpyToSymbol(HIP_SYMBOL(d_filter_const_), filter_, sizeof(float) * filterSize_, NULL, hipMemcpyHostToDevice));
}

// copy data from host to device including filter & image data
void GFilter::prepareMemory(float *imgIn, int wid, int hei) {
    //hipError_t cudaState = hipSuccess;
    cudaErrorCheck(hipMemcpy(d_imgIn_, imgIn, sizeof(float) * wid * hei, hipMemcpyHostToDevice));
}

// do gaussian filtering on global memory
void GFilter::gaussfilterGlo(float *imgOut, float *imgIn, int wid, int hei, float *filter, int filterW) {

    createfilter();
    prepareMemory(imgIn, wid, hei);

    dim3 threadPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 blockPerGrid;

    blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (hei + threadPerBlock.y - 1) / threadPerBlock.y;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    gaussfilterGlo_kernel << < blockPerGrid, threadPerBlock >> > (d_imgOut_, d_imgIn_, wid, hei, d_filter_, filterW_);
    //gaussfilterCon_kernel<<<blockPerGrid, threadPerBlock>>>(d_imgOut_, d_imgIn_, wid, hei, d_filter_, filterW_);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "GPU Time = :" << elapsedTime << " ms" << endl;

    cudaErrorCheck(hipMemcpy(imgOut, d_imgOut_, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));

    cout << "In gaussfilterGlo Function :" << hipGetErrorString(hipPeekAtLastError()) << endl;
}

void GFilter::gaussfilterTex(float *imgOut, float *imgIn, int wid, int hei, float *filter, int filterW) {
    createfilter();

    size_t pitch;

    float *d_imgIn_Pitch;
    cudaErrorCheck(hipMallocPitch((void **) &d_imgIn_Pitch, &pitch, wid * sizeof(float), hei));

    // copy image data from host to 2D Pitch
    cudaErrorCheck(hipMemcpy2D(d_imgIn_Pitch, pitch, imgIn, wid * sizeof(float), wid * sizeof(float), hei,
                                hipMemcpyHostToDevice));

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    texIn.addressMode[0] = texIn.addressMode[1] = hipAddressModeBorder;
    //texIn.addressMode[0] = hipAddressModeBorder;
    //texIn.addressMode[1] = hipAddressModeBorder;
    // bind the texture to 2D Pitch
    cudaErrorCheck(hipBindTexture2D(NULL, texIn, d_imgIn_Pitch, channelDesc, wid, hei, pitch));

    dim3 threadPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 blockPerGrid;

    blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (hei + threadPerBlock.y - 1) / threadPerBlock.y;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    gaussfilterTex_kernel <<< blockPerGrid, threadPerBlock >>>
                                             (d_imgOut_, d_imgIn_Pitch, wid, hei, d_filter_, filterW_);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "GPU Time = :" << elapsedTime << " ms" << endl;

    hipUnbindTexture(texIn);

    cudaErrorCheck(hipMemcpy(imgOut, d_imgOut_, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));
}

void GFilter::gaussfilterSha(float *imgOut, float *imgIn, int wid, int hei, float *filter, int filterW) {
    createfilter();
    prepareMemory(imgIn, wid, hei);


    dim3 threadPerBlock(BLOCKSIZE, BLOCKSIZE);
    dim3 blockPerGrid;
    blockPerGrid.x = (wid + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (hei + threadPerBlock.y - 1) / threadPerBlock.y;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    gaussfilterSha_kernel<<< blockPerGrid, threadPerBlock>>>(d_imgOut_, d_imgIn_, wid, hei, d_filter_, filterW);

    cout << hipGetErrorString(hipPeekAtLastError()) << endl;

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "GPU Time = :" << elapsedTime << " ms" << endl;

    cudaErrorCheck(hipMemcpy(imgOut, d_imgOut_, sizeof(float) * wid * hei, hipMemcpyDeviceToHost));
}

void GFilter::gaussfilterSep(float *imgOut, float *imgIn, int wid, int hei, float *filter, int filterW)
{
}

void GFilter::gaussfilterShaSep(float *imgOut, float *imgIn, int wid, int hei, float *filter, int filterW)
{

}
