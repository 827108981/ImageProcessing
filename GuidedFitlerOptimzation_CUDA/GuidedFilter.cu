#include "hip/hip_runtime.h"
//
// Author : smh - 2017.01.16
//

#include "GuidedFilter.h"
#include "hip/hip_vector_types.h"

#define BLOCKSIZE 32   // BLOCKSIZE * BLOCKSIZE threads per block

using namespace std;
using namespace cv;

texture<float4, hipTextureType2D> rgbaTex;
hipArray *rgbaIn_d, *rgbaOut_d;

int iDiv(int a, int b)
{
    if (a % b == 0)
        return a / b;
    else
        return a / b + 1;
}

void imgShow(Mat img)
{
    imshow("Temp", img);
    waitKey(0);
}

GFilter::GFilter(int r, int c) : row_(r), col_(c), rad_(45), eps_(0.000001)
{
    hipEventCreate(&startEvent_);
    hipEventCreateWithFlags(&stopEvent_, hipEventBlockingSync);
}

GFilter::~GFilter()
{
    hipEventDestroy(startEvent_);
    hipEventDestroy(stopEvent_);
}

// Kernel functions
// __device__
// do boxfilter
__global__ void
d_boxfilter_rgb_x(float4* d_out, int row, int col, int rad)
{
    float scale = 1.0f / (float)((rad << 1) + 1.0f);
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

    if (y < row)
    {
        float4 t = make_float4(0.0f);
        for (int x = -rad; x <= rad; ++x)
        {
            t += tex2D(rgbaTex, x, y);
        }

        d_out[y * col] = t * scale;

        for (int x = 1; x < col; ++x)
        {
            t += tex2D(rgbaTex, x + rad, y);
            t -= tex2D(rgbaTex, x - rad - 1, y);
            d_out[y * col + x] = t * scale;
        }
    }
}

// CAUTION: The input is transposed in Texture Memory ! ! !
__global__ void
testTexture(float4 *d_out, int row, int col)
{
    unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

    if( y < row )
    {
        float4 t = make_float4(0.0f);
        for (int x = 0; x < col; ++x)
        {
            t = tex2D(rgbaTex, x, y);

            d_out[x * row + y] = t;
        }
    }
}

__global__ void
d_boxfilter_rgb_y(float4* d_out_, float4* d_in_, const int row, const int col, const int rad)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < col)
    {
        float4 *d_in = &d_in_[x];
        float4 *d_out = &d_out_[x];

        float scale = 1.0f / (float)((rad << 1) + 1.0f);

        float4 t = make_float4(0.0f);

        t = d_in[0] * rad;

        for (int y = 0; y < (rad + 1); y++)
        {
            t += d_in[y * col];
        }

        d_out[0] = t * scale;

        // do up edge
        for (int y = 1; y < rad + 1; y++)
        {
            t += d_in[(y + rad) * col];
            t -= d_in[0];
            d_out[y * col] = t * scale;
        }

        // do main loop
        for (int y = (1 + rad); y < (row - rad); y++)
        {
            t += d_in[(y + rad) * col];
            t -= d_in[(y - rad) * col];
            d_out[y * col] = t * scale;
        }

        // do right edge
        for (int y = row - rad; y < row; y++)
        {
            t += d_in[(row - 1) * col];
            t -= d_in[((y - rad) * col) - col];

            d_out [y * col] = t * scale;
        }
    }
}

void GFilter::initTexture(float* data)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float* tempH = new float [row_ * col_ * 4];
    float* tempSrc = tempH;
    float* tempD = data;
    const int size = row_ * col_;
    for (int i = 0; i < size; ++i)
    {
        *tempH++ = *tempD++;
        *tempH++ = *tempD++;
        *tempH++ = *tempD++;
        *tempH++ = 0.0;
    }

    // allocate the 2d Array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
    //hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    cudaCheckError(hipMallocArray(&rgbaIn_d, &channelDesc, col_, row_));

    hipEventRecord(start);
    cudaCheckError(hipMemcpyToArray(rgbaIn_d, 0, 0, tempSrc, size * sizeof(float4), hipMemcpyHostToDevice));
    // Until here, It is correct ! ! !

    // bind array to texture
    cudaCheckError(hipBindTextureToArray(rgbaTex, rgbaIn_d, channelDesc));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Copy Data + Bind Texture: " << elapsedTime << " ms." << endl;

    delete [] tempSrc;
}

void releaseTexture()
{
    hipUnbindTexture(rgbaTex);
    cudaCheckError(hipFreeArray(rgbaIn_d));
    //cudaCheckError(hipFreeArray(rgbaOut_d));
}

void GFilter::restoreFromFloat4(float *out, float *in)
{
    float *tempIn = in;
    float *tempOut = out;

    for (int i = 0; i < row_; ++i)
        for (int j = 0; j < col_; ++j)
        {
            *tempOut++ = *tempIn++;
            *tempOut++ = *tempIn++;
            *tempOut++ = *tempIn++;
            ++tempIn;
        }
}

void GFilter::boxfilter(float *imgOut_d, const float *imgIn_d, int rad)
{
}

void GFilter::boxfilterTest(cv::Mat &imgOut, const cv::Mat &imgIn, int rad)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *dataInP = (float *)imgIn.data;
    float *dataOutP = (float *)imgOut.data;

    float4 *tempData, *outDataD;
    float *tempDataH = new float [row_ * col_ * sizeof(float4)];
    //hipChannelFormatDesc channels = hipCreateChannelDesc<float4>();
    cudaCheckError(hipMalloc((void **)&tempData, sizeof(float4) * col_ * row_));
    cudaCheckError(hipMalloc((void **)&outDataD, sizeof(float4) * col_ * row_));

    initTexture(dataInP);
    hipEventRecord(start);
    // use texture for horizontal pass
    //dim3 blockPerGrid = ;
    d_boxfilter_rgb_x<<<iDiv(row_, BLOCKSIZE), BLOCKSIZE, 0>>>(tempData, row_, col_, rad);   // use row_ / BLOCKSIZE, because the input is transposed.
    //d_boxfilter_rgb_x<<<iDiv(row_, BLOCKSIZE), BLOCKSIZE, 0>>>(outDataD, row_, col_, rad);
    //testTexture<<<row_ / BLOCKSIZE, BLOCKSIZE, 0>>>(outDataD, row_, col_);     // The Result is transposed of input matrix
    d_boxfilter_rgb_y<<<iDiv(col_, BLOCKSIZE), BLOCKSIZE, 0>>>(outDataD, tempData, row_, col_, rad);
    //cout << hipGetErrorString(hipPeekAtLastError()) << endl;

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime = 0.0;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Boxfilter on GPU(no data transfer: " << elapsedTime << " ms." << endl;

    cudaCheckError(hipMemcpy(tempDataH, outDataD, sizeof(float4) * row_ * col_, hipMemcpyDeviceToHost));
    //cudaCheckError(hipMemcpy(tempDataH, tempData, sizeof(float4) * row_ * col_, hipMemcpyDeviceToHost));
    //cudaCheckError(hipMemcpyFromArray(tempDataH, rgbaIn_d, 0, 0, sizeof(float4) * row_ * col_, hipMemcpyDeviceToHost));    // CORRECT ! ! !

    restoreFromFloat4(dataOutP, tempDataH);

    delete [] tempDataH;
    releaseTexture();
}

void GFilter::boxfilterNpp(cv::Mat &imgOut, const cv::Mat &imgIn, int rad)
{
    assert(imgIn.isContinuous());
    const float* imgI_h = (const float*)imgIn.data;
    float* imgOut_h = (float *)imgOut.data;
    int pSrcStepBytes = col_ * sizeof(float) * imgIn.channels();

    int pStepBytes;
    Npp32f* imgIn_d = nppiMalloc_32f_C3(col_, row_, &pStepBytes);
    NppStatus stateNpp = NPP_SUCCESS;
    hipError_t stateCUDA = hipSuccess;
    NppiSize sizeROI;
    sizeROI.width = col_;
    sizeROI.height = row_;
    // Copy image from host to device
    stateCUDA = hipMemcpy2D(imgIn_d, pStepBytes, imgI_h, pSrcStepBytes, pSrcStepBytes, row_, hipMemcpyHostToDevice);
    assert(stateCUDA == hipSuccess);
    Npp32f* imgOut_d = nppiMalloc_32f_C3(col_, row_, &pStepBytes);
    NppiSize oMaskSize = {16, 16};
    NppiPoint oAnchor = {oMaskSize.width/2, oMaskSize.height / 2};

    hipEventRecord(startEvent_, 0);
    stateNpp = nppiFilterBoxBorder_32f_C3R(imgIn_d, pStepBytes, sizeROI, {0,0}, imgOut_d, pStepBytes, sizeROI, oMaskSize, oAnchor, NPP_BORDER_REPLICATE);
    hipEventRecord(stopEvent_, 0);
    //hipEventSynchronize(stopEvent_);
    hipEventElapsedTime(&elapsedTime_, startEvent_, stopEvent_);
    cout << "Only GPU Time: " << elapsedTime_ << "ms." << endl;
    if (stateNpp != NPP_SUCCESS)
    {
        nppiFree(imgIn_d);
        nppiFree(imgOut_d);
        exit(EXIT_FAILURE);
    }

    stateCUDA = hipMemcpy2D(imgOut_h, pSrcStepBytes, imgOut_d, pStepBytes, pStepBytes, row_, hipMemcpyDeviceToHost);
    assert(stateCUDA == hipSuccess);
    hipDeviceSynchronize();
    nppiFree(imgIn_d);
    nppiFree(imgOut_d);
}

void GFilter::gaussianfilter(float *imgOut_d, const float *imgIn_d, int rad, double sig)
{
}

// 输入图像是相同的  e.g. imgInI == imgInP
// color image guided filter
void GFilter::guidedfilterSingle(cv::Mat &imgOut, const cv::Mat &imgInI, const cv::Mat &imgInP)
{
}

// 输入图像是不同的  e.g. imgInI != imgInP
void GFilter::guidedfilterDouble(cv::Mat &imgOut, const cv::Mat &imgInI, const cv::Mat &imgInP)
{
}

void GFilter::guidedfilter(cv::Mat &imgOut, const cv::Mat &imgInI, const cv::Mat &imgInP)
{
    assert(imgInP.channels() == 3 && imgInI.channels() == 3);
    //const float *imgA = (float *)imgInI.data;
    //const float *imgB = (float *)imgInP.data;
    equal_to<const float*> T;
    if (T((float *)imgInI.data, (float*)imgInP.data))
        guidedfilterSingle(imgOut, imgInI, imgInI);
    else
        guidedfilterDouble(imgOut, imgInI, imgInP);
}

// Contrast Experiments: Guided Filter based on OpenCV
void GFilter::guidedfilterOpenCV(cv::Mat &imgOut, const cv::Mat &imgInI, const cv::Mat &imgInP)
{
    assert(imgInP.channels() == 3);
    if (rad_ == 0)
        setParams(16, 0.01);    // Image Enhancement

    Mat meanI, corrI, varI, meanP;
    boxFilter(imgInI, meanI, imgInI.depth(), Size(rad_, rad_));
    boxFilter(imgInI.mul(imgInI), corrI, imgInI.depth(), Size(rad_, rad_));
    boxFilter(imgInP, meanP, imgInP.depth(), Size(rad_, rad_));
    varI = corrI - meanI.mul(meanI);
    //imgShow(varI);

    vector<Mat> vecP(imgInP.channels()), vecI(imgInI.channels());
    vector<Mat> vecMeanI(imgInI.channels()), vecMeanP(imgInP.channels());
    split(imgInP, vecP);
    split(imgInI, vecI);
    split(meanP, vecMeanP);
    split(meanI, vecMeanI);

    Mat covIp, sameP, sameMeanP, meanA, meanB;
    vector<Mat> vecA(imgInI.channels());
#pragma unloop
    for (int i = 0; i < 3; ++i)
    {
        //vector<Mat> vecSameP{vecP[i], vecP[i], vecP[i]};
        //merge(vecSameP, sameP);
        //boxFilter(imgInI.mul(sameP), covIp, imgInI.depth(), Size(rad_, rad_));
        //vector<Mat> vecSameMeanP{vecMeanP[i], vecMeanP[i], vecMeanP[i]};
        //merge(vecSameMeanP, sameMeanP);
        cvtColor(vecP[i], sameP, CV_GRAY2BGR);         // use cvtColor to do the broadcast purpose, instead of above method
        cvtColor(vecMeanP[i], sameMeanP, CV_GRAY2BGR);
        boxFilter(imgInI.mul(sameP), covIp, imgInI.depth(), Size(rad_, rad_));
        covIp = covIp - meanI.mul(sameMeanP);

        Mat a = covIp / (varI + eps_);
        boxFilter(a, meanA, a.depth(), Size(rad_, rad_));
        //cout << "a.channels = " << a.channels() << endl;         // for test

        split(a, vecA);
        Mat b = vecMeanP[i] - (vecA[0].mul(vecMeanI[0]) + vecA[1].mul(vecMeanI[1]) + vecA[2].mul(vecMeanI[2]));
        boxFilter(b, meanB, b.depth(), Size(rad_, rad_));
        //cout << "b.channels = " << b.channels() << endl;         // for test

        split(meanA, vecA);
        vecP[i] = vecA[0].mul(vecI[0]) + vecA[1].mul(vecI[1]) + vecA[2].mul(vecI[2]) + meanB;
    }
    merge(vecP, imgOut);
}
