#include "hip/hip_runtime.h"
#include "boxfilter.h"

texture<float, hipTextureType2D> texIn;

__global__ void boxfilter_kernel(float *out, int wid, int hei, const size_t pitch, const int fWid, const int fHei)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;

	float outputVal = 0.0f;

	if(idx < wid && idy < hei)
	{
		for(int i = -fWid; i <= fWid; ++i)
		{
			for(int j = -fHei; j <= fHei; ++j)
				outputVal += tex2D(texIn, idx + i, idy + j);
		}
		outputVal /= ((2 * fWid + 1) * (2 * fHei + 1));

		int offset = idy * pitch / sizeof(float) + idx;
		out[offset] = outputVal;
	}
}

// 2D Array Memory Version
void BFilter::boxfilter()
{
    hipError_t cudaState = hipSuccess;

    size_t pitch = width * sizeof(float);

    hipChannelFormatDesc channelDescArray = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray *cudaArrayIn;
    cudaState = hipMallocArray(&cudaArrayIn, &channelDescArray, width, height, hipArrayDefault);
    assert(cudaState == hipSuccess);
    cudaState = hipMalloc((void **)&dataOutD, width * height * sizeof(float));
    assert(cudaState == hipSuccess);

    // copy data from host to hipArray
    cudaState = hipMemcpyToArray(cudaArrayIn, 0, 0, data, width * height * sizeof(float), hipMemcpyHostToDevice);
    assert(cudaState == hipSuccess);

    // set texture reference parameters
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    texIn.addressMode[0] = texIn.addressMode[1] = hipAddressModeBorder;
    cudaState = hipBindTextureToArray(texIn, cudaArrayIn, channelDesc);
    assert(cudaState == hipSuccess);

    // launch the kernel
    dim3 threadPerBlock(16, 16);
    dim3 blockPerGrid;
    blockPerGrid.x = (width + threadPerBlock.x - 1) / threadPerBlock.x;
    blockPerGrid.y = (height + threadPerBlock.y - 1) / threadPerBlock.y;

    boxfilter_kernel<<<blockPerGrid, threadPerBlock>>>(dataOutD, width, height, pitch, rad, rad);

    //cudaState = hipMemcpyToArray(data, 0, 0, cudaArrayIn, width * height * sizeof(float), hipMemcpyDefault);
    // copy data back to host
    cudaState = hipMemcpy(data, dataOutD, width * height * sizeof(float), hipMemcpyDeviceToHost);
    assert(cudaState == hipSuccess);

    // Unbind the texture
    hipFreeArray(cudaArrayIn);  // there is no need to call hipUnbindTexture ... no such API
}

// 2D Linear memory version
/*
void BFilter::boxfilter()
{
	hipError_t cudaState = hipSuccess;

	// The pitch is the width in bytes of the allocation
	size_t pitch = 0;
	cudaState = hipMallocPitch((void **)&dataInD, &pitch, width * sizeof(float), height);   // the requested pitched allocation width is in bytes !!!
	assert(cudaState == hipSuccess);
	cudaState = hipMallocPitch((void **)&dataOutD, &pitch, width * sizeof(float), height);
	assert(cudaState == hipSuccess);

	cout << "Pitch = " << pitch << endl;

	// copy data from 2D host memory to device memory
	cudaState = hipMemcpy2D(dataInD, pitch, data, width * sizeof(float), width * sizeof(float), height, hipMemcpyHostToDevice);    // the spitch and width is also in bytes !!!
	assert(cudaState == hipSuccess);

	// bind the dataInD to texture memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	texIn.addressMode[0] = texIn.addressMode[1] = hipAddressModeBorder;
	cudaState = hipBindTexture2D(0, texIn, dataInD, channelDesc,width, height, pitch);    // the width & height is in texel units, but pitch is in bytes
	assert(cudaState == hipSuccess);

	// launch the kernel
	dim3 threadPerBlock(16, 16);
	dim3 blockPerGrid;
	blockPerGrid.x = (width + threadPerBlock.x - 1) / threadPerBlock.x;
	blockPerGrid.y = (height + threadPerBlock.y - 1) / threadPerBlock.y;

	boxfilter_kernel<<<blockPerGrid, threadPerBlock>>>(dataOutD, width, height, pitch, rad, rad);

	// copy data back to host
	hipMemcpy2D(data, width * sizeof(float), dataOutD, pitch, width * sizeof(float), height, hipMemcpyDeviceToHost);

	// Unbind the texture memory
	cudaState = hipUnbindTexture(texIn);
	assert(cudaState == hipSuccess);
}
*/


void BFilter::print()
{
    for(int i = 0; i < height; ++i)
    {
        for(int j = 0; j < width; ++j)
        {
            if(j < width - 1)
                std::cout << data[j + i * width] << ", ";
            else
                std::cout << data[j + i * width];
        }
        std::cout << "; " << std::endl;
    }
}

BFilter::~BFilter()
{
    if(!dataInD)
        hipFree(dataInD);
    if(!dataOutD)
        hipFree(dataOutD);
}
