#include "hip/hip_runtime.h"
#include <iostream>
#include "cassert"
#include "vector"

#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

using namespace std;
using namespace cv;

#define TILE_DIM 32
#define BLOCK_ROWS 8

__global__ void
dataCopy(float *out, float *in, int row, int col)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx < col && idy < row)
    {
        int offset = idy * col + idx;
        out[offset] = in[offset];
    }
}

__global__ void
transposeNaive(float *out, float *in, int row, int col)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx < col && idy < row)
    {
        int inIdx = idy * col + idx;
        int outIdx = idx * row + idy;
        out[outIdx] = in[inIdx];
    }
}

__global__ void
transposeShareMem(float *out, float *in, int row, int col)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx < col && idy < row)
    {
        int inIdx = idy * col + idx;
    }
}

int main() {
    std::cout << "Hello, World!" << std::endl;
    Mat imgIn = imread("barbara.jpg", IMREAD_GRAYSCALE);
    assert(!imgIn.empty());
    imgIn.convertTo(imgIn, CV_32F, 1.0/255);

    Mat imgOut = Mat::zeros(imgIn.size(), CV_32F);

    const float *imgInP = (float *)imgIn.data;
    float *imgOutP = (float *)imgOut.data;
    const int row = imgIn.rows;
    const int col = imgIn.cols;

    hipError_t cudaState = hipSuccess;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime = 0.0;

    float *imgInD, *imgOutD;
    size_t pitchSrc, pitchDst;
    cudaState = hipMallocPitch((void **)&imgInD, &pitchSrc, col * sizeof(float), row);   // the width is in bytes
    assert(cudaState == hipSuccess);
    cudaState = hipMemcpy2D(imgInD, pitchSrc, imgInP, sizeof(float) * col, sizeof(float) * col, row, hipMemcpyHostToDevice);
    assert(cudaState == hipSuccess);
    cudaState = hipMallocPitch((void **)&imgOutD, &pitchDst, col * sizeof(float), row);
    assert(cudaState == hipSuccess);

    dim3 threadPerBlock(TILE_DIM, TILE_DIM);
    dim3 blockPerGrid;
    blockPerGrid.x = (col + TILE_DIM - 1) / TILE_DIM;
    blockPerGrid.y = (row + TILE_DIM - 1) / TILE_DIM;

    hipEventRecord(start);
    //dataCopy<<<blockPerGrid, threadPerBlock>>>(imgOutD, imgInD, row, col);              // 0.063648ms
    //transposeNaive<<<blockPerGrid, threadPerBlock>>>(imgOutD, imgInD, row, col);          // 0.196096ms

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Used Time: " << elapsedTime << " ms." << endl;


    cudaState = hipMemcpy2D(imgOutP, sizeof(float) * col, imgOutD, pitchDst, sizeof(float) * col, row, hipMemcpyDeviceToHost);
    assert(cudaState == hipSuccess);

    imshow("Output", imgOut);
    waitKey(0);

    return 0;
}
